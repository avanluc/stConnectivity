#include "hip/hip_runtime.h"
#pragma once

#include <assert.h>
#include "GlobalSync.cu"
#include "GlobalWrite.cu"
#include "CacheFunc.cu"



/*
* kERNEL FUNCTION THAT CHECKS IF EVERY NODE HAS BEEN VISITED
*/
__global__ void CheckVisit(	const int* __restrict__ devNode, 
							const int* __restrict__ devEdge,
							int* __restrict__ BitMask,
							const int BMsize,
							const int N)
{
	int founds = 0;
	for (int i = GTid; i < N; i+=MAX_CONCURR_TH)
		if(markAccess<hipcub::LOAD_CS, int>(BitMask, i) == 0 ){
			founds++;
	}

	atomicAdd(&VisitResult, founds);
}


/*
* KERNEL FUNCTION FOR SET EXTRA BITS TO 1
*/
__global__ void initBitMask(int* BitMask, const int N, const int MaskSize)
{
	int totBits = MaskSize * 8 * sizeof(int);
	if(GTid < (totBits - N))
		markWrite<hipcub::LOAD_CS, hipcub::STORE_CS, int>(BitMask, N + GTid);
}



/*
* KERNEL FUNCTION FOR STCONNECTIVITY
*/
__global__ void STCONN_BlockKernel (const int* __restrict__ devNode,
									const int* __restrict__ devEdge,
									const int* __restrict__ devSource,
									int2* __restrict__ devDistance,
									bool* __restrict__ Matrix,
									int* __restrict__ BitMask, 
									const int Nsources,
									const int E)
{
	int Queue[REG_QUEUE];
	int FrontierSize = 1;
	int level = 0;
	int* SMemF1 = (int*) &SMem[F1_OFFSET];
	int* F2SizePtr = (int*) &SMem[F2Size_POS];

	
	for(int j = Bid; j < Nsources; j += gridDim.x)
	{
		if (Tid < FrontierSize){
			SMemF1[Tid] = devSource[j];
			markWrite<hipcub::LOAD_CG, hipcub::STORE_CG, int>(BitMask, SMemF1[Tid]);
			//AtomicMarkWrite(BitMask, SMemF1[Tid]);
		}
	
		while ( FrontierSize && FrontierSize < BLOCK_FRONTIER_LIMIT )
		{
			int founds = 0; int counter = 0;
			for (int t = Tid; t < FrontierSize; t += BLOCK_SIZE)
			{
				const int index = SMemF1[t];
				const int start = devNode[index];
				const int2 current = devDistance[index];
				const int end = devNode[index + 1];	
	
				for (int k = start; k < end; k++)
				{
					const int dest = devEdge[k];
					const int2 destination = devDistance[dest];	
	
					if(founds < REG_QUEUE)
					{
						counter++;
						if ( atomicCAS(&devDistance[dest].x, INT_MAX, level) == INT_MAX ) {
							devDistance[dest].y = current.y;
							Queue[founds++] = dest;
							//AtomicMarkWrite(BitMask, dest);
							markWrite<hipcub::LOAD_CG, hipcub::STORE_CG, int>(BitMask, dest);
						}
						else if (destination.y != current.y && destination.y < Nsources){	
							Matrix[ (current.y     * Nsources) + destination.y ] = true;	
							Matrix[ (destination.y * Nsources) + current.y 	   ] = true;	
						}
					}
				}
			}

			Write(SMemF1, &F2SizePtr[0], Queue, founds);
	
			level++;
			FrontierSize = F2SizePtr[0];
			GlobalWrite(counter, &GlobalCounter);

			__syncthreads();
			F2SizePtr[0] = 0;

			if(__int2double_rn(GlobalCounter) / __int2double_rn(E) > TRESHOLD)
				return;
		}
		FrontierSize = 1;  level = 0; 
	}
}



/*
* BOTTOM UP KERNEL
*/
__global__ void Bottom_Up_Kernel(	const int* __restrict__ devNode,
									const int* __restrict__ devEdge,
									int*  __restrict__ BitMask,
									const int BMsize,
									const int N)
{
	const int stride = gridDim.x * BLOCK_SIZE * 8;
	int* BitMarkArray = BitMask + Tid * 4;
	int founds  = 0;
	for (int BlockIndex = Bid * BLOCK_SIZE * 8; BlockIndex < BMsize; BlockIndex += stride)
	{
		char Queue[32];

		reinterpret_cast<int4*>(Queue)[0] = reinterpret_cast<int4*>(BitMarkArray + BlockIndex)[0];
		reinterpret_cast<int4*>(Queue)[1] = __ldg( &reinterpret_cast<int4*>(BitMarkArray + BlockIndex)[BLOCK_SIZE] );

		#pragma unroll
		for (int i = 0; i < 32; i++){
			#pragma unroll
			for (int j = 0; j < 8; j++){
				const int ldg_stride = i >= 16 ? BLOCK_SIZE * 128 : 0;
				const int index = (BlockIndex * 32) + (Tid * 128) + ldg_stride + (i%16 * 8) + j;
				if ((Queue[i] & (1 << (7 - j))) == 0 && index < N && visitAdjiacentBit(index, devNode, devEdge, BitMask))
				{
					Queue[i] |= (1 << (7 - j));
					founds++;
				}
			}
		}

		reinterpret_cast<int4*>(BitMarkArray + BlockIndex)[0] = reinterpret_cast<int4*>(Queue)[0];
		reinterpret_cast<int4*>(BitMarkArray + BlockIndex)[BLOCK_SIZE] = reinterpret_cast<int4*>(Queue)[1];
		
		BitMarkArray += stride;
	}
	GlobalWrite( founds, &BottomUp_FrontSize);
}
