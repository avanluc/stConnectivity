#include "hip/hip_runtime.h"
#pragma once

#include <../../hipcub/hipcub.hpp>

__device__ unsigned int GSync[MAX_CONCURR_BL(BLOCK_SIZE)];
__device__ bool devNextLevel[4];

__global__ void GReset() {
	int bx = blockIdx.x;
    int tx = threadIdx.x;
    int id = tx + (bx*BLOCK_SIZE);
	if (id < MAX_CONCURR_BL(BLOCK_SIZE))
		GSync[id] = 0;
}

/*
*	Glocal Syncronization function
*/
__device__  __forceinline__ void GlobalSync() {
	volatile unsigned *VolatilePtr = GSync;
	__syncthreads();
	
	if (blockIdx.x == 0) {
		if (threadIdx.x == 0){
			VolatilePtr[blockIdx.x] = 1;
		}
		//__syncthreads();

		if (threadIdx.x < MAX_CONCURR_BL(BLOCK_SIZE))
			while ( hipcub::ThreadLoad<hipcub::LOAD_CG>(GSync + threadIdx.x) == 0 );

		__syncthreads();

		if (threadIdx.x < MAX_CONCURR_BL(BLOCK_SIZE)){
			VolatilePtr[threadIdx.x] = 0;
		}
	}
	else {
		if (threadIdx.x == 0) {
			VolatilePtr[blockIdx.x] = 1;
			while (hipcub::ThreadLoad<hipcub::LOAD_CG>(GSync + blockIdx.x) == 1);
		}
		__syncthreads();
	}
}