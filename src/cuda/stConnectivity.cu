#include "hip/hip_runtime.h"
#include <KernelStConnectivity_WEfficient.cuh>

using namespace std;


int main(int argc, char *argv[]){

	// If wrong number of argumants print usage
	if(argc < 3)
	{
		printf("\nUsage: ./stConnectivity 'input_file' '#_distinguished_nodes'\n\n");
		return -1;
	}

	// Read graph from file
	edge *graph;
	ifstream in (argv[1]);
	in >> N >> E;
	graph = (edge*)malloc(E*sizeof(edge));
	ReadGraph(argv[1], graph);
	
	// Memory allocation and initialization
	int *vertex = (int*)calloc(N+1, sizeof(int));
	int *edges = (int*)calloc(E, sizeof(int));
	
	// Creation of CSR structure
	GraphToCSR(graph, vertex, edges);
	printf("Transfert graph to CSR structure\n");
	
	// int source = atoi(argv[2]);
	// int target = atoi(argv[3]);
	// int Nsources = atoi(argv[4]);

	int Nsources = atoi(argv[2]);

	// calculate size for allocation
	size_t sizeE 	= E * sizeof(int);
	//size_t sizeN 	= N * sizeof(int);
	size_t sizeN 	= N * sizeof(int2);
	size_t sizeN1 	= (N+1) * sizeof(int);
	size_t sizeN3 	= Nsources * sizeof(int);
	size_t sizeMatrix = Nsources * Nsources * sizeof(bool);

	// Allocate adj matrix in row major
	// int *Color 		= (int*)calloc(N, sizeof(int));
	// int *Distance 	= (int*)calloc(N, sizeof(int));	
	int2 *Distance 	= (int2*)calloc(N, sizeof(int2));	
	int *sources 	= (int*)calloc(Nsources, sizeof(int));
	int *Queue 		= (int*)calloc(Nsources, sizeof(int));	
	bool *matrix 	= (bool*)calloc(Nsources * Nsources, sizeof(bool));

	// Allocate device memory
	int *Dedges;
	int *Dvertex;
	int *Dsources;
	int2 *Ddistance;
	// int *Ddistance;
	// int *Dcolor;
	bool *DMatrix;

	gpuErrchk( hipMalloc((void **) &Dvertex, 	sizeN1) );
	gpuErrchk( hipMalloc((void **) &Dedges, 	sizeE) );
	gpuErrchk( hipMalloc((void **) &DMatrix, 	sizeMatrix) );
	gpuErrchk( hipMalloc((void **) &Ddistance, sizeN) );
	//gpuErrchk( hipMalloc((void **) &Dcolor, 	sizeN) );
	gpuErrchk( hipMalloc((void **) &Dsources, 	sizeN3) );
	printf("Device memory allocated\n");
	printf("-----------------------------------\n");

 	vector<double> mean_times(3);
 	vector<double> par_times(N_TEST);
 	vector<double> seq_times(N_TEST);

	srand (time(NULL));
 	for (int test = 0; test < N_TEST; test++)
	{

		int source = rand() % N;
		int target = rand() % N;

		// choose Nsources distinguished nodes with source and target in it and return a vector of them	
	    ChooseNodes(sources, vertex, N, Nsources, source, target);

	    // inizializzazione dei valori a INT_MAX
	    for (int i = 0; i < N; i++){
	    	Distance[i].x = INT_MAX;
	    	Distance[i].y = INT_MAX;
	  		// Distance[i] = INT_MAX;
			// Color[i] = INT_MAX;
	    }

	    // inizializzazione dei valori dei nodi distinti a Distance 0 e Color id
	    for (int i = 0; i < Nsources; i++){
			int j = sources[i];
			Distance[j].x = 0;
			Distance[j].y = i;
			// Distance[j] = 0;
			// Color[j] = i;
		}
		Distance[0].x = 0;
		
	    // Copy host memory for vertex, edges and results vectors to device
	    gpuErrchk( hipMemcpy(Dvertex, vertex, sizeN1, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(DMatrix, matrix, sizeMatrix, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dedges, edges, sizeE, hipMemcpyHostToDevice) );
	    //gpuErrchk( hipMemcpy(Dcolor, Color, sizeN, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Ddistance, Distance, sizeN, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dsources, sources, sizeN3, hipMemcpyHostToDevice) );
	    
		// Allocate CUDA events to be used for timing
	    hipEvent_t start;
	    hipEvent_t start1;
	    hipEvent_t stop;
	    hipEvent_t stop1;

	    gpuErrchk( hipEventCreate(&start) );
	    gpuErrchk( hipEventCreate(&start1) );
	    gpuErrchk( hipEventCreate(&stop) );
	    gpuErrchk( hipEventCreate(&stop1) );
	    
	    // Record the start event
	    gpuErrchk( hipEventRecord(start, NULL) );

	    // Launch Cuda Kernel
		dim3 block(BLOCK_SIZE, 1);
	    dim3 grid(1, 1);
	    
	    BFS_BlockKernel<<< grid, block, SM_BYTE_PER_BLOCK>>>(Dvertex, Dedges, Dsources, Ddistance, /*Dcolor,*/ DMatrix, Nsources);

		gpuErrchk( hipMemcpy(matrix, DMatrix, sizeMatrix, hipMemcpyDeviceToHost) );
		// for (int i = 0; i < Nsources; ++i)
		// {
		// 	printf("| ");
		// 	for (int j = 0; j < Nsources; ++j)
		// 		printf("%d ", matrix[Nsources*i+j]);
		// 	printf("|\n");
		// }
		//printf("matrix completed\n");
		
	    gpuErrchk( hipEventRecord(stop, NULL) );
	    gpuErrchk( hipEventSynchronize(stop) );
		gpuErrchk( hipEventRecord(start1, NULL) );
	    
	    bool connect = false;
	    connect = MatrixBFS(matrix, Nsources, 0, 1, Queue);

		gpuErrchk( hipEventRecord(stop1, NULL) );
	    gpuErrchk( hipEventSynchronize(stop1) );

	    // Claculate elapsed time
	    float msecTotal = 0.0f;
	    float msecTotal1 = 0.0f;
	    gpuErrchk( hipEventElapsedTime(&msecTotal, start, stop) );
	    gpuErrchk( hipEventElapsedTime(&msecTotal1, start1, stop1) );

	    //if(!DEBUG){
			printf("#%d:\tst-Connectivity from %d\t   to %d\tis %c[%d;%dm%s%c[%dm\t\tElapsed time = %c[%d;%dm%.1f%c[%dm ms\n", 
															test, source, target, 27, 0, 31 + connect,(connect ? "true" : "false"), 
															27, 0, 27, 0, 31, msecTotal + msecTotal1, 27, 0);
			//printf("Parallel Time : %c[%d;%dm%.1f%c[%dm ms\n", 27, 0, 31, msecTotal, 27, 0);

			//printf("-----------------------------------\n");	
	    //}
		par_times[test] = msecTotal;
		seq_times[test] = msecTotal1;
	}
	double sum_par = 0;
	double sum_seq = 0;
	for (int i = 1; i < N_TEST; ++i){
		sum_par += par_times[i];
		sum_seq += seq_times[i];
	}
	printf("\nN: %d\n", Nsources);
	printf("AVG TIME \t\t: %c[%d;%dm%.1f%c[%dm ms\n", 27, 0, 31, (sum_par + sum_seq) / (N_TEST-1), 27, 0);
	printf("AVG PARALLEL TIME \t: %c[%d;%dm%.1f%c[%dm ms\n", 27, 0, 31, sum_par / (N_TEST-1), 27, 0);
	printf("AVG MATRIX BFS TIME \t: %c[%d;%dm%.1f%c[%dm ms\n\n", 27, 0, 31, sum_seq / (N_TEST-1), 27, 0);

	//free memory
	hipFree(Dvertex);
    hipFree(Dedges);
    hipFree(DMatrix);
    //hipFree(Dcolor);
    hipFree(Ddistance);

    free(Queue);
	free(matrix);
	//free(Color);
	free(Distance);
	free(vertex);
	free(edges);
	free(graph);

	return 0;
}