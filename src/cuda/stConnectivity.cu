#include "hip/hip_runtime.h"
#include <KernelStConnectivity_WEfficient.cuh>

using namespace std;


int main(int argc, char *argv[]){

	// If wrong number of argumants print usage
	if(argc < 3)
	{
		printf("\nUsage: ./stConnectivity 'input_file' '#_distinguished_nodes'\n\n");
		return -1;
	}

	// Read graph from file
	edge *graph;
	ifstream in (argv[1]);
	in >> N >> E;
	graph = (edge*)malloc(E*sizeof(edge));
	ReadGraph(argv[1], graph);
	
	// Memory allocation and initialization
	int *vertex = (int*)calloc(N+1, sizeof(int));
	int *edges = (int*)calloc(E, sizeof(int));
	
	// Creation of CSR structure
	GraphToCSR(graph, vertex, edges);
	//printf("Transfert graph to CSR structure\n");
	
	// int source = atoi(argv[2]);
	// int target = atoi(argv[3]);
	// int nof_distNodes = atoi(argv[4]);
	
	// // Source/Target control
	// if( source >= N )
	// {
	// 	printf("Source node > |N| = %d\n", N);
	// 	return -1;
	// }
	// if( target >= N )
	// {
	// 	printf("Target node > |N| = %d\n", N);
	// 	return -1;	
	// }

	int nof_distNodes = atoi(argv[2]);

	// calculate size for allocation
	size_t sizeN 	= (N+1) * sizeof(int);
	size_t sizeE 	= E * sizeof(int);
	size_t sizeN2 	= N * sizeof(int2);
	size_t sizeN3 	= nof_distNodes * sizeof(int);
	size_t sizeMATRIX = nof_distNodes * nof_distNodes * sizeof(bool);

	// Allocate adj matrix in row major
	bool *matrix 	= (bool*)calloc(nof_distNodes * nof_distNodes, sizeof(bool));
	int2 *Dist_Col 	= (int2*)calloc(N, sizeof(int2));
	int *Distance 	= (int*)calloc(nof_distNodes, sizeof(int));	
	int *sources 	= (int*)calloc(nof_distNodes, sizeof(int));
	int *Queue 		= (int*)calloc(nof_distNodes, sizeof(int));	

	// Allocate device memory
	int *Dedges;
	int *Dvertex;
	int *Ddistance;
	int *Dsources;
	int2 *Ddist_Col;
	bool *DMatrix;
	//bool *DnewLevel;

	gpuErrchk( hipMalloc((void **) &Dvertex, sizeN) );
	gpuErrchk( hipMalloc((void **) &Ddistance, sizeN3) );
	gpuErrchk( hipMalloc((void **) &Dedges, sizeE) );
	gpuErrchk( hipMalloc((void **) &DMatrix, sizeMATRIX) );
	gpuErrchk( hipMalloc((void **) &Ddist_Col, sizeN2) );
	gpuErrchk( hipMalloc((void **) &Dsources, sizeN3) );
	//printf("Device memory allocated\n");
	//printf("-----------------------------------\n");

 	vector<double> mean_times(3);
 	vector<double> par_times(N_TEST);
 	vector<double> seq_times(N_TEST);

	srand (time(NULL));
 	for (int test = 0; test < N_TEST; test++)
	{

		int source = rand() % N;
		int target = rand() % N;

		// choose nof_distNodes distinguished nodes with source and target in it and return a vector of them	
	    ChooseNodes(sources, vertex, N, nof_distNodes, source, target);

	    // inizializzazione dei valori a INT_MAX
	    for (int i = 0; i < N; i++)
	    {
	    	Dist_Col[i].x = INT_MAX;
			Dist_Col[i].y = INT_MAX;
	    }

	    // inizializzazione dei valori dei nodi distinti a Distance 0 e Color id
	    for (int i = 0; i < nof_distNodes; i++)
		{
			int j = sources[i];
			Dist_Col[j].x = 0;
			Dist_Col[j].y = i;
			Distance[i] = INT_MAX;
		}
		Distance[0] = 0;
		//bool NextLevel[4] = {0,0,0,0};
		
	    // Copy host memory for vertex, edges and results vectors to device
	    gpuErrchk( hipMemcpy(Dvertex, vertex, sizeN, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dedges, edges, sizeE, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(DMatrix, matrix, sizeMATRIX, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Ddist_Col, Dist_Col, sizeN2, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Ddistance, Distance, sizeN3, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dsources, sources, sizeN3, hipMemcpyHostToDevice) );
	    //gpuErrchk( hipMemcpyToSymbolAsync(HIP_SYMBOL(devNextLevel), NextLevel, sizeof(bool)*4, 0,hipMemcpyHostToDevice) );

		// Allocate CUDA events to be used for timing
	    hipEvent_t start;
	    hipEvent_t start1;
	    hipEvent_t stop;
	    hipEvent_t stop1;

	    gpuErrchk( hipEventCreate(&start) );
	    gpuErrchk( hipEventCreate(&start1) );
	    gpuErrchk( hipEventCreate(&stop) );
	    gpuErrchk( hipEventCreate(&stop1) );
	    
	    // Record the start event
	    gpuErrchk( hipEventRecord(start, NULL) );

	    // Launch Cuda Kernel
		dim3 block(BLOCK_SIZE, 1);
	    //dim3 grid(MAX_CONCURR_BL(BLOCK_SIZE), 1);
	    dim3 grid(1, 1);
	    
	    //GReset<<< grid, block >>>();
	    
	    //stConn<<< grid, block >>>(Dvertex, Dedges, Ddist_Col, N, nof_distNodes, DMatrix);
	    BFS_BlockKernel<<< grid, block, SM_BYTE_PER_BLOCK>>>(Dvertex, Dedges, Ddist_Col, Dsources, nof_distNodes);


		// Print matrix
		// gpuErrchk( hipMemcpy(matrix, DMatrix, sizeMATRIX, hipMemcpyDeviceToHost) );
		// for (int i = 0; i < nof_distNodes; ++i)
		// {
		// 	printf("| ");
		// 	for (int j = 0; j < nof_distNodes; ++j)
		// 		printf("%d ", matrix[nof_distNodes*i+j]);
		// 	printf("|\n");
		// }
		//printf("matrix completed\n");
		
	    gpuErrchk( hipEventRecord(stop, NULL) );
	    gpuErrchk( hipEventSynchronize(stop) );

	    bool connect = false;
	    if(!DEBUG){
		    gpuErrchk( hipEventRecord(start1, NULL) );

		    // Copy result vector from device to host
			gpuErrchk( hipMemcpy(matrix, DMatrix, sizeMATRIX, hipMemcpyDeviceToHost) );
			connect = MatrixBFS(matrix, nof_distNodes, 0, 1, Queue);
			gpuErrchk( hipEventRecord(stop1, NULL) );
		    gpuErrchk( hipEventSynchronize(stop1) );
	    }


	    // Claculate elapsed time
	    float msecTotal = 0.0f;
	    float msecTotal1 = 0.0f;
	    gpuErrchk( hipEventElapsedTime(&msecTotal, start, stop) );
	    gpuErrchk( hipEventElapsedTime(&msecTotal1, start1, stop1) );

	    if(!DEBUG){
			printf("#%d:\tst-Connectivity from %d\t   to %d\tis %c[%d;%dm%s%c[%dm\t\tElapsed time = %c[%d;%dm%.1f%c[%dm ms\n", 
															test, source, target, 27, 0, 31 + connect,(connect ? "true" : "false"), 
															27, 0, 27, 0, 31, msecTotal + msecTotal1, 27, 0);
			printf("-----------------------------------\n");	
	    }
		par_times[test] = msecTotal;
		seq_times[test] = msecTotal1;
	}
	double sum_par = 0;
	double sum_seq = 0;
	for (int i = 1; i < N_TEST; ++i){
		sum_par += par_times[i];
		sum_seq += seq_times[i];
	}
	if(!DEBUG){
		printf("\nN: %d\n", nof_distNodes);
		printf("AVG TIME \t\t: %c[%d;%dm%.1f%c[%dm ms\n", 27, 0, 31, (sum_par + sum_seq) / (N_TEST-1), 27, 0);
		printf("AVG PARALLEL TIME \t: %c[%d;%dm%.1f%c[%dm ms\n", 27, 0, 31, sum_par / (N_TEST-1), 27, 0);
		printf("AVG MATRIX BFS TIME \t: %c[%d;%dm%.1f%c[%dm ms\n\n", 27, 0, 31, sum_seq / (N_TEST-1), 27, 0);
	}

	//free memoria device
	hipFree(Dvertex);
    hipFree(Dedges);
    hipFree(DMatrix);
    hipFree(Ddist_Col);
    hipFree(Ddistance);

    free(Queue);
	free(matrix);
	free(Dist_Col);
	free(Distance);
	free(vertex);
	free(edges);
	free(graph);

	return 0;
}