#include "hip/hip_runtime.h"
//#pragma once

#include "Kernel_STCONN.cu"
#include "stConn.h"
#include "statistic.h"



void doSTCONN(Graph graph, int N, int E, double Treshold){
	
	/***    CALCULATE SIZES    ***/
	size_t sizeE 	  = E * sizeof(int);
	size_t sizeN 	  = N * sizeof(int2);
	size_t sizeSrcs	  = 2 * sizeof(int);
	size_t sizeBMask  = N * sizeof(bool);
	size_t sizeN1 	  = (N+1) * sizeof(int);
	size_t sizeMatrix = MAX_SIZE * MAX_SIZE * sizeof(bool);
	size_t sizeCurand = MAX_CONCURR_BL(BLOCK_SIZE) * sizeof(hiprandState);
	

	/***    ALLOCATE HOST MEMORY    ***/
	int2 *Distance 	= (int2*)calloc(N, sizeof(int2));	
	int *sources 	= (int*)calloc(2, sizeof(int));
	int *Queue 		= (int*)calloc(MAX_SIZE, sizeof(int));	
	bool *matrix 	= (bool*)calloc(MAX_SIZE * MAX_SIZE, sizeof(bool));
	

	/***    ALLOCATE DEVICE MEMORY    ***/
	int *Dedges;
	int *Dvertex;
	int *Dsource;
	bool *DBitMask;
	int2 *Ddistance;
	bool *DMatrix;
	hiprandState* devStates;


	/***    SERVICE VARIABLES    ***/
	std::vector<double> mean_times(3);
	std::vector<double> par_times(N_TEST);
	std::vector<double> BOT_times(N_TEST);
	std::vector<double> seq_times(N_TEST);
	std::vector<long double> Percentual(N_TEST);
	int        totSrc = 0;
	int    connectCnt = 0;
	int   percCounter = 0;
	int unfinishedCnt = 0;
	long double  perc = 0.0;


	srand (time(NULL));
	for (int test = 0; test < N_TEST; ++test)
	{

	    gpuErrchk( hipMalloc((void **) &devStates, sizeCurand) );
		gpuErrchk( hipMalloc((void **) &Dvertex, 	sizeN1) );
		gpuErrchk( hipMalloc((void **) &Dedges, 	sizeE) );
		gpuErrchk( hipMalloc((void **) &DMatrix, 	sizeMatrix) );
		gpuErrchk( hipMalloc((void **) &Ddistance, sizeN) );
		gpuErrchk( hipMalloc((void **) &Dsource, 	sizeSrcs) );
		gpuErrchk( hipMalloc((void **) &DBitMask, 	sizeBMask) );



		/***    CHOOSE RANDOM SOURCE AND DEST    ***/
		int source = rand() % N;
		int target = rand() % N;
		while(target == source)		target = rand() % N;


		/***    STRUCTURES INITIALIZATION    ***/
		for (int i = 0; i < N; ++i)
			Distance[i] = make_int2(INT_MAX, INT_MAX);

		sources[0] = source;
		sources[1] = target;
		Distance[source] = make_int2(0, 0);
		Distance[target] = make_int2(0, 1);



		/***    MEMCOPY HOST_TO_DEVICE    ***/
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(VisitResult), &ZERO, sizeof(int), 0, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(VisitResult1), &ZERO, sizeof(int), 0, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(GlobalCounter), &ZERO, sizeof(int), 0, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(color), &INIT_COLOR, sizeof(int), 0, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(Dvertex, graph.OutNodes, sizeN1, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(Dedges, graph.OutEdges, sizeE, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(Ddistance, Distance, sizeN, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(Dsource, sources, sizeSrcs, hipMemcpyHostToDevice) );		
		gpuErrchk( hipMemset(DMatrix, false, sizeMatrix) );
		gpuErrchk( hipMemset(DBitMask, false, sizeBMask) );

		
		/***    INITIALIZE TIMERS    ***/
		Timer<HOST> TM;
		Timer<HOST> TM1;
		Timer<DEVICE> TM_TD;
		Timer<DEVICE> TM_BU;
		float msecPAR = 0.0f;
		float msecBOT = 0.0f;
		float msecSEQ = 0.0f;
		bool  connect = false;
		int  tempSources = 0;
		int VisitedEdges = 0;



		/***    LAUNCH RESET KERNEL    ***/
		clean<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>();
		setup_curand<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>(devStates);
	


		/***    LAUNCH STCONN TOP-DOWN KERNEL    ***/
		TM_TD.start();
		TopDown_Kernel<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>\
					(Dvertex, Dedges, Dsource, Ddistance, DMatrix, DBitMask, Treshold, E, N, devStates);
		TM_TD.stop();


		int a = 0; int b = 0;
		CheckVisit<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>(DBitMask, Ddistance, N);
		gpuErrchk( hipMemcpyFromSymbol(&a, HIP_SYMBOL(VisitResult), sizeof(int), 0, hipMemcpyDeviceToHost) );
		gpuErrchk( hipMemcpyFromSymbol(&b, HIP_SYMBOL(VisitResult1), sizeof(int), 0, hipMemcpyDeviceToHost) );
		if(a != b)
		{
			printf("\t!!! There are %d nodes without color !!!\n", a );
			printf("\t!!! There are %d nodes not visited !!!\n", b);
		}
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(VisitResult), &ZERO, sizeof(int), 0, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(VisitResult1), &ZERO, sizeof(int), 0, hipMemcpyHostToDevice) );


		/***    CHECK VISIT PERCENTAGE AND SOURCES    ***/
		gpuErrchk( hipMemcpyFromSymbol(&tempSources, HIP_SYMBOL(color), sizeof(int), 0, hipMemcpyDeviceToHost) );
		gpuErrchk( hipMemcpyFromSymbol(&VisitedEdges, HIP_SYMBOL(GlobalCounter), sizeof(int), 0, hipMemcpyDeviceToHost) );
		perc = ((long double)VisitedEdges / (long double)E) * 100.0;
		totSrc += (tempSources-1);
		


		if(BOTTOM_UP && perc >= Treshold*100 )
		{
			int result = 0;
			int result1 = 0;
			int FrontierSize = 1;



			/***    LAUNCH STCONN BOTTOM_UP KERNEL    ***/
			TM_BU.start();
			while( FrontierSize )
			{
				gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(BottomUp_FrontSize), &ZERO, sizeof(int), 0, hipMemcpyHostToDevice) );
				BottomUp_Kernel<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>\
								(Dvertex, Dedges, Ddistance, DMatrix, DBitMask, N);
				gpuErrchk( hipMemcpyFromSymbol(&FrontierSize, HIP_SYMBOL(BottomUp_FrontSize), sizeof(int), 0, hipMemcpyDeviceToHost) );
			}
			TM_BU.stop();



			/***    CHECK IF THE VISIT IS COMPLETE    ***/
			CheckVisit<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>(DBitMask, Ddistance, N);

			gpuErrchk( hipMemcpyFromSymbol(&result, HIP_SYMBOL(VisitResult), sizeof(int), 0, hipMemcpyDeviceToHost) );
			gpuErrchk( hipMemcpyFromSymbol(&result1, HIP_SYMBOL(VisitResult1), sizeof(int), 0, hipMemcpyDeviceToHost) );

			if(result1)
				printf("\t!!! There are %d nodes without color !!!\n", result1 );
			if(result)
				printf("\t!!! There are %d nodes not visited !!!\n", result);
		}
		else
		{
			printf("---------------WARNING: TRESHOLD NOT REACHED---------------\t\t %d on %d\t%.2Lf%%\n", VisitedEdges, E, perc);
			Percentual[percCounter++] = perc;
		}



		/***    MATRIX STCONN ON HOST    ***/
		TM.start();		    
		gpuErrchk( hipMemcpy(matrix, DMatrix, sizeMatrix, hipMemcpyDeviceToHost) );
		connect = MatrixBFS(matrix, MAX_SIZE, 0, 1, Queue);
		TM.stop();	    	
		
		//PrintMatrix<bool>(matrix, MAX_SIZE);

		msecPAR = TM_TD.duration();
		msecBOT = TM_BU.duration();
		msecSEQ = TM.duration();


		PrintResults(test, source, target, connect, msecPAR + msecBOT + msecSEQ);


		#if SEQ_CHECK
			bool connect1 = stConnectivity(graph.OutNodes, graph.OutEdges, N, source, target);
			PrintResults(test, source, target, connect1, -1.0);
		#endif



		/***    SAVE TIMES FOR STATISTICS EVALUATION    ***/
		par_times[test] = msecPAR;
		seq_times[test] = msecSEQ;
		BOT_times[test] = msecBOT;

		if( perc < 100 )
			unfinishedCnt++;


		/***    FREE DEVICE MEMORY    ***/
		hipFree(Ddistance);
		hipFree(DBitMask);
		hipFree(Dsource);
		hipFree(DMatrix);
		hipFree(Dvertex);
		hipFree(Dedges);
		hipFree(devStates);
	}
	


	/***    EVALUATE MEAN TIMES AND PERCENTAGE    ***/
	computeElapsedTime( par_times, seq_times, BOT_times, connectCnt);
	printf("AVG SOURCES        \t: %d\n", totSrc/N_TEST);
	computeMeanPercentage(Percentual, percCounter);
	


	/***    FREE HOST MEMORY    ***/
	free(Distance);
	free(matrix);
	free(Queue);
	free(sources);
}



/*
* Read command line parameters
*/
void Parameters(int argc, char* argv[], GDirection &GDir, double& Treshold) {
    std::string errString(
    "Syntax Error:\n\n stConnectivity <graph_path> [ <graph_direction> ] [ -n <number_of_sources>] [-A]\n\n\
    <graph_direction>:\n\
                    -D      force directed graph\n\
                    -U      force undirected graph");

    if (argc < 2)
        error(errString)
    GDir = UNDEFINED;
    for (int i = 2; i < argc; ++i)
    {
        std::string parameter = argv[i];

        if 		(parameter.compare("-D") == 0)
            GDir = DIRECTED;
        else if (parameter.compare("-U") == 0)
            GDir = UNDIRECTED;
        else if ( /*i + 1 < argc &&*/ parameter.compare("-n") == 0 && 
        		std::string(argv[i + 1]).find_first_not_of("0123456789.") == std::string::npos )
        {
            std::istringstream ss(argv[++i]);
            ss >> Treshold;
        }
        else
            error(errString)
    }
}



/*
* Main function
*/
int main(int argc, char *argv[]){

	/***    READ GRAPH FROM FILE    ***/
	int N, E, nof_lines;
	double Treshold = 0.0;
	GDirection GraphDirection;		//DIRECTED = 0, UNDIRECTED = 1, UNDEFINED = 2
	Parameters(argc, argv, GraphDirection, Treshold);
	readGraph::readGraphHeader(argv[1], N, E, nof_lines, GraphDirection);
	Graph graph(N, E, GraphDirection);
	readGraph::readSTD(argv[1], graph, nof_lines);
	graph.DegreeAnalisys();


	/***    PRINT CONFIG INFO    ***/
	std::cout << "\n----------------------KERNEL INFO---------------------" 			<< std::endl
		 << "            Block dimension : " <<  BLOCK_SIZE 							<< std::endl
		 << "      Max concurrent blocks : " <<  MAX_CONCURR_BL(BLOCK_SIZE) 			<< std::endl
		 << "       Shared Memory per SM : " <<  SMem_Per_SM 							<< std::endl
		 << "    Shared Memory per block : " <<  SMem_Per_Block(BLOCK_SIZE) 			<< std::endl
		 << "Int Shared Memory per block : " <<  IntSMem_Per_Block(BLOCK_SIZE) 			<< std::endl
		 << "         Frontier dimension : " <<  FRONTIER_SIZE 							<< std::endl
		 << "         Int frontier limit : " <<  BLOCK_FRONTIER_LIMIT 					<< std::endl
		 << "--------------------------------------------------------" 	   << std::endl << std::endl;

	/***    LAUNCH ST-CONN FUNCTION    ***/
	if(Treshold != 0)
	{
		printf("\n----------Launch stConnectivity with treshold %.2f%%----------\n\n", Treshold*100);
		doSTCONN(graph, N, E, Treshold);
	}
	else
	{
		for (int i = 0; i < LENGTH; ++i)
		{
			printf("\n----------Launch stConnectivity with treshold %.2f%%----------\n\n", TRESH[i]*100);
			doSTCONN(graph, N, E, TRESH[i]);
		}
	}
	return 0;
}