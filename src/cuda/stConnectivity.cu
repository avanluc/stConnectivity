#include "hip/hip_runtime.h"
//#pragma once

#include "Kernel_STCONN.cu"
#include "stConn.h"
#include "statistic.h"


void doSTCONN(Graph graph, int N, int E, double Treshold){
	
	/***    CALCULATE SIZES    ***/
	size_t sizeE 	  = E * sizeof(int);
	size_t sizeN 	  = N * sizeof(int2);
	size_t sizeN1 	  = (N+1) * sizeof(int);
	size_t sizeSrcs	  = 2 * sizeof(int);
	size_t sizeMatrix = MAX_SIZE * MAX_SIZE * sizeof(bool);
	size_t sizeBMask  = N * sizeof(bool);
	size_t sizeCurand = MAX_CONCURR_BL(BLOCK_SIZE) * sizeof(hiprandState);
	//size_t sizeBMask  = (N + (1024 * BLOCK_SIZE)) * sizeof(bool);
	

	/***    ALLOCATE HOST MEMORY    ***/
	int2 *Distance 	= (int2*)calloc(N, sizeof(int2));	
	int *sources 	= (int*)calloc(2, sizeof(int));
	int *Queue 		= (int*)calloc(MAX_SIZE, sizeof(int));	
	bool *matrix 	= (bool*)calloc(MAX_SIZE * MAX_SIZE, sizeof(bool));
	

	/***    ALLOCATE DEVICE MEMORY    ***/
	int *Dedges;
	int *Dvertex;
	int *Dsource;
	bool *DBitMask;
	int2 *Ddistance;
	bool *DMatrix;
	hiprandState* devStates;

    gpuErrchk( hipMalloc((void **) &devStates, sizeCurand) );
	gpuErrchk( hipMalloc((void **) &Dvertex, 	sizeN1) );
	gpuErrchk( hipMalloc((void **) &Dedges, 	sizeE) );
	gpuErrchk( hipMalloc((void **) &DMatrix, 	sizeMatrix) );
	gpuErrchk( hipMalloc((void **) &Ddistance, sizeN) );
	gpuErrchk( hipMalloc((void **) &Dsource, 	sizeSrcs) );
	gpuErrchk( hipMalloc((void **) &DBitMask, 	sizeBMask) );


	/***    SERVICE VARIABLES    ***/
	std::vector<double> mean_times(3);
	std::vector<double> par_times(N_TEST);
	std::vector<double> BOT_times(N_TEST);
	std::vector<double> seq_times(N_TEST);
	std::vector<long double> Percentual(N_TEST);
	int 	connectCnt = 0;
	int    percCounter = 0;
	int  unfinishedCnt = 0;
	long double   perc = 0.0;


	srand (time(NULL));
	for (int test = 0; test < N_TEST; ++test)
	{

		/***    CHOOSE RANDOM SOURCE, DEST AND EXTRA-SOURCES    ***/
		int VisitedEdges = 0;
		int source = rand() % N;
		int target = rand() % N;
		while(target == source)		target = rand() % N;


		/***    STRUCTURES INITIALIZATION    ***/
		for (int i = 0; i < N; ++i)
			Distance[i] = make_int2(INT_MAX, INT_MAX);

		sources[0] = source;
		sources[1] = target;
		Distance[source] = make_int2(0, 0);
		Distance[target] = make_int2(0, 1);


		/***    MEMCOPY HOST_TO_DEVICE    ***/
		gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(GlobalCounter), &VisitedEdges, sizeof(int)) );
		gpuErrchk( hipMemcpy(Dvertex, graph.OutNodes, sizeN1, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(Dedges, graph.OutEdges, sizeE, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(DMatrix, matrix, sizeMatrix, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(Ddistance, Distance, sizeN, hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(Dsource, sources, sizeSrcs, hipMemcpyHostToDevice) );		
		gpuErrchk( hipMemset(DBitMask, 0, sizeBMask) );

		
		/***    INITIALIZE TIMERS    ***/
		Timer<HOST> TM;
		Timer<DEVICE> TM_TD;
		Timer<DEVICE> TM_BU;
		float msecPAR = 0.0f;
		float msecSEQ = 0.0f;
		float msecBOT = 0.0f;
		bool  connect = false;


		/***    LAUNCH RESET KERNEL    ***/
		setup_curand<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>(devStates);

			int totSouce = 0;
			
		/***    LAUNCH STCONN TOP-DOWN KERNEL    ***/
		TM_TD.start();
		TopDown_Kernel<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>\
					(Dvertex, Dedges, Dsource, Ddistance, DMatrix, DBitMask, Treshold, E, N, devStates);
		TM_TD.stop();
		msecPAR = TM_TD.duration();

			hipMemcpyFromSymbol(&totSouce, HIP_SYMBOL(color), sizeof(int), 0, hipMemcpyDeviceToHost);
			printf("Total sources = %d\n", totSouce);

		/***    CHECK VISIT PERCENTAGE    ***/
		gpuErrchk( hipMemcpyFromSymbol(&VisitedEdges, HIP_SYMBOL(GlobalCounter), sizeof(int), 0, hipMemcpyDeviceToHost) );
		perc = ((long double)VisitedEdges / (long double)E) * 100.0;
		

		if(BOTTOM_UP && perc > Treshold*100 )
		{

			//int FrontierSize1 = 0;
			//int level = 0;
			int FrontierSize = 1;
			int zero = 0;
			int result = 0;
			int BMsize = ceil((double)N / 4.0);

			TM_BU.start();
			while( FrontierSize )
			{
				//gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(BottomUp_FrontSize1), &zero, sizeof(int), 0, hipMemcpyHostToDevice) );
				gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(BottomUp_FrontSize), &zero, sizeof(int), 0, hipMemcpyHostToDevice) );
				BottomUp_Kernel<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>\
								(Dvertex, Dedges, DBitMask, BMsize, N);
				gpuErrchk( hipMemcpyFromSymbol(&FrontierSize, HIP_SYMBOL(BottomUp_FrontSize), sizeof(int), 0, hipMemcpyDeviceToHost) );
				//gpuErrchk( hipMemcpyFromSymbol(&FrontierSize1, HIP_SYMBOL(BottomUp_FrontSize1), sizeof(int), 0, hipMemcpyDeviceToHost) );
			}
			TM_BU.stop();
			msecBOT = TM_BU.duration();



			hipMemcpyToSymbol(HIP_SYMBOL(VisitResult), &zero, sizeof(int), 0, hipMemcpyHostToDevice);
			CheckVisit<<< MAX_CONCURR_BL(BLOCK_SIZE), BLOCK_SIZE, SMem_Per_Block(BLOCK_SIZE)>>>(DBitMask, N);
			hipMemcpyFromSymbol(&result, HIP_SYMBOL(VisitResult), sizeof(int), 0, hipMemcpyDeviceToHost);
			if(result != 0)
				printf("\t!!! There are %d nodes not visited !!!\n", result);
			
		}
		else
		{
			printf("---------------WARNING: TRESHOLD NOT REACHED---------------\t\t %d on %d\t%.2Lf%%\n", VisitedEdges, E, perc);
			Percentual[percCounter++] = perc;
		}


		/***    MATRIX STCONN ON HOST    ***/
		#if !BFS
			TM.start();		    
			
			//gpuErrchk( hipMemcpy(matrix, DMatrix, sizeMatrix, hipMemcpyDeviceToHost) );
			//connect = MatrixBFS(matrix, MAX_SIZE, 0, 1, Queue);

			TM.stop();	    	
			msecSEQ = TM.duration();
		#endif


		/***    PRINT RESULTS    ***/
		#if (!BFS)
			printf("#%d:\tsource: %d     \ttarget: %d      \tresult: %c[%d;%dm%s%c[%dm\t\ttime = %c[%d;%dm%.1f%c[%dm ms\n", 
															test+1, source, target, 27, 0, 31 + connect,(connect ? "true" : "false"), 
															27, 0, 27, 0, 31, msecPAR + msecSEQ + msecBOT, 27, 0);
		#endif


		/***    SAVE TIMES FOR STATISTICS EVALUATION    ***/
		par_times[test] = msecPAR;
		seq_times[test] = msecSEQ;
		BOT_times[test] = msecBOT;

		if( perc < 100 )
			unfinishedCnt++;
	}
	

	/***    EVALUATE MEAN TIMES AND PERCENTAGE    ***/
	computeElapsedTime( par_times, seq_times, BOT_times, connectCnt);
	computeMeanPercentage(Percentual, percCounter);
	
	/***    FREE DEVICE MEMORY    ***/
	hipFree(Ddistance);
	hipFree(Dvertex);
	hipFree(DMatrix);
	hipFree(Dedges);
	hipFree(DBitMask);
	hipFree(Dsource);

	/***    FREE HOST MEMORY    ***/
	free(Distance);
	free(matrix);
	free(Queue);
	free(sources);
}



/*
* Read command line parameters
*/
void Parameters(int argc, char* argv[], GDirection &GDir, double& Treshold) {
    std::string errString(
    "Syntax Error:\n\n stConnectivity <graph_path> [ <graph_direction> ] [ -n <number_of_sources>] [-A]\n\n\
    <graph_direction>:\n\
                    -D      force directed graph\n\
                    -U      force undirected graph");

    if (argc < 2)
        error(errString)
    GDir = UNDEFINED;
    for (int i = 2; i < argc; ++i)
    {
        std::string parameter = argv[i];

        if 		(parameter.compare("-D") == 0)
            GDir = DIRECTED;
        else if (parameter.compare("-U") == 0)
            GDir = UNDIRECTED;
        else if ( /*i + 1 < argc &&*/ parameter.compare("-n") == 0 && 
        		std::string(argv[i + 1]).find_first_not_of("0123456789.") == std::string::npos )
        {
            std::istringstream ss(argv[++i]);
            ss >> Treshold;
        }
        else
            error(errString)
    }
}



/*
* Main function
*/
int main(int argc, char *argv[]){

	/***    READ GRAPH FROM FILE    ***/
	int N, E, nof_lines;
	double Treshold = 0.0;
	GDirection GraphDirection;		//DIRECTED = 0, UNDIRECTED = 1, UNDEFINED = 2
	Parameters(argc, argv, GraphDirection, Treshold);
	readGraph::readGraphHeader(argv[1], N, E, nof_lines, GraphDirection);
	Graph graph(N, E, GraphDirection);
	readGraph::readSTD(argv[1], graph, nof_lines);
	graph.DegreeAnalisys();


	/***    PRINT CONFIG INFO    ***/
	std::cout << "\n----------------------KERNEL INFO---------------------" 			<< std::endl
		 << "            Block dimension : " <<  BLOCK_SIZE 							<< std::endl
		 << "      Max concurrent blocks : " <<  MAX_CONCURR_BL(BLOCK_SIZE) 			<< std::endl
		 << "       Shared Memory per SM : " <<  SMem_Per_SM 							<< std::endl
		 << "    Shared Memory per block : " <<  SMem_Per_Block(BLOCK_SIZE) 			<< std::endl
		 << "Int Shared Memory per block : " <<  IntSMem_Per_Block(BLOCK_SIZE) 			<< std::endl
		 << "         Frontier dimension : " <<  FRONTIER_SIZE 							<< std::endl
		 << "         Int frontier limit : " <<  BLOCK_FRONTIER_LIMIT 					<< std::endl
		 << "--------------------------------------------------------" 	   << std::endl << std::endl;

	/***    LAUNCH ST-CONN FUNCTION    ***/
	if(Treshold != 0)
	{
		printf("\n----------Launch stConnectivity with treshold %.2f%%----------\n\n", Treshold*100);
		doSTCONN(graph, N, E, Treshold);
	}
	else
	{
		for (int i = 0; i < LENGTH; ++i)
		{
			printf("\n----------Launch stConnectivity with treshold %.2f%%----------\n\n", Treshold*100);
			doSTCONN(graph, N, E, Treshold);
		}
	}
	return 0;
}