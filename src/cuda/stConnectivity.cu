#include "hip/hip_runtime.h"
#pragma once

#include "Kernel_StConnectivity.cu"
#include "stConn.h"


void doSTCONN(Graph graph, int N, int E, int Nsources){
	
	/***    CALCULATE SIZES    ***/
	size_t sizeE 	  = E * sizeof(int);
	size_t sizeN 	  = N * sizeof(int2);
	size_t sizeN1 	  = (N+1) * sizeof(int);
	size_t sizeSrcs	  = Nsources * sizeof(int);
	size_t sizeMatrix = Nsources * Nsources * sizeof(bool);

	

	/***    ALLOCATE HOST MEMORY    ***/
	int2 *Distance 	= (int2*)calloc(N, sizeof(int2));	
	int *sources 	= (int*)calloc(Nsources, sizeof(int));
	int *Queue 		= (int*)calloc(Nsources, sizeof(int));	
	bool *matrix 	= (bool*)calloc(Nsources * Nsources, sizeof(bool));


	/***    ALLOCATE DEVICE MEMORY    ***/
	int *Dedges;
	int *Dvertex;
	int *Dsources;
	int2 *Ddistance;
	bool *DMatrix;

	gpuErrchk( hipMalloc((void **) &Dvertex, 	sizeN1) );
	gpuErrchk( hipMalloc((void **) &Dedges, 	sizeE) );
	gpuErrchk( hipMalloc((void **) &DMatrix, 	sizeMatrix) );
	gpuErrchk( hipMalloc((void **) &Ddistance, sizeN) );
	gpuErrchk( hipMalloc((void **) &Dsources, 	sizeSrcs) );


	/***    SERVICE VARIABLES    ***/
 	std::vector<double> mean_times(3);
 	std::vector<double> par_times(N_TEST);
 	std::vector<double> seq_times(N_TEST);
 	std::vector<long double> Percentual(N_TEST);
 	int connectCnt = 0;
 	int percCounter = 0;
 	int unfinishedCnt = 0;
 	long double perc = 0.0;


	srand (time(NULL));
 	for (int test = 0; test < N_TEST; test++)
	{

		/***    CHOOSE RANDOM SOURCE, DEST AND EXTRA-SOURCES    ***/
		int source = rand() % N;
		int target = rand() % N;

	    ChooseRandomNodes(sources, graph.OutNodes, N, Nsources, source, target);


	    /***    STRUCTURES INITIALIZATION    ***/
	    for (int i = 0; i < N; i++){
	    	Distance[i].x = INT_MAX;
	    	Distance[i].y = INT_MAX;
	    }

	    for (int i = 0; i < Nsources; i++){
			int j = sources[i];
			Distance[j].x = 0;
			Distance[j].y = i;
		}
		
		int VisitedNodes = 0;
	    
	    /***    MEMCOPY HOST_TO_DEVICE    ***/
	    gpuErrchk( hipMemcpy(Dvertex, graph.OutNodes, sizeN1, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dedges, graph.OutEdges, sizeE, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(DMatrix, matrix, sizeMatrix, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Ddistance, Distance, sizeN, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dsources, sources, sizeSrcs, hipMemcpyHostToDevice) );		
		if(ATOMIC)
			gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(GlobalCounter), &VisitedNodes, sizeof(int)) );
	    
		
		/***    ALLOCATE CUDA EVENT FOR TIMING    ***/
	    hipEvent_t start;
	    hipEvent_t stop;

	    gpuErrchk( hipEventCreate(&start) );
	    gpuErrchk( hipEventCreate(&stop) );
	    gpuErrchk( hipEventRecord(start, NULL) );


		/***    LAUNCH KERNEL    ***/
		dim3 block(BLOCK_SIZE, 1);
	    dim3 grid(Nsources, 1);
	    BFS_BlockKernel<<< grid, block, SMem_Per_Block(BLOCK_SIZE)>>>(Dvertex, Dedges, Dsources, Ddistance, DMatrix, Nsources);


	    /***    MEMCOPY DEVICE_TO_HOST    ***/
	    if(!BFS)
			gpuErrchk( hipMemcpy(matrix, DMatrix, sizeMatrix, hipMemcpyDeviceToHost) );


	    /***    RECORD STOP TIME    ***/
	    gpuErrchk( hipEventRecord(stop, NULL) );
	    gpuErrchk( hipEventSynchronize(stop) );


	    /***    COPY EXITFLAG FROM DEVICE    ***/
	    if(!ATOMIC){
		    int Flag = 0;
		    gpuErrchk( hipMemcpyFromSymbol(&Flag, HIP_SYMBOL(exitFlag), sizeof(int), 0, hipMemcpyDeviceToHost) );
		    if(Flag){
		    	unfinishedCnt = N_TEST - test;
		    	break;
		    }
	    }


	    /***    CHECK VISIT PERCENTAGE IF IT FAILS    ***/
	    if(ATOMIC){
			gpuErrchk( hipMemcpyFromSymbol(&VisitedNodes, HIP_SYMBOL(GlobalCounter), sizeof(int), 0, hipMemcpyDeviceToHost) );
			VisitedNodes += Nsources;
			perc = ((long double)VisitedNodes / (long double)N) * 100;
			if(VisitedNodes < N){
				printf("---------------WARNING: BFS NOT COMPLETE---------------\t\t\t\t%.2Lf%\n", perc);
				Percentual[percCounter] = perc;
				percCounter++;
			}
		}


	    float msecTotal = 0.0f;
	    float msecTotal1 = 0.0f;
	    bool connect = false;


	    /***    MATRIX VISIT ON HOST    ***/
	    if(!BFS){
		    Timer<HOST> TM;
		    TM.start();		    
			connect = MatrixBFS(matrix, Nsources, 0, 1, Queue);
			TM.stop();	    	
		    msecTotal1 = TM.duration();
	    }


		/***    CALCULATE ELAPSED TIME    ***/
	    gpuErrchk( hipEventElapsedTime(&msecTotal, start, stop) );
	    
	    if(DEBUG)
			printf("#%d:\tsource: %d     \ttarget: %d      \tresult: %c[%d;%dm%s%c[%dm   \tElapsed time = %c[%d;%dm%.1f%c[%dm ms\n", 
															test, source, target, 27, 0, 31 + connect,(connect ? "true" : "false"), 
															27, 0, 27, 0, 31, msecTotal + msecTotal1, 27, 0);
		par_times[test] = msecTotal;
		seq_times[test] = msecTotal1;
		
		if( !connect )
			connectCnt++;

	    if( ATOMIC && perc < 100 )
			unfinishedCnt++;
	}
	

	/***    EVALUATE MEAN TIMES    ***/
	if(N_TEST > 1)
		computeElapsedTime( par_times, seq_times, connectCnt);


	/***    EVALUATE MEAN PERCENTAGE    ***/
	if(ATOMIC)
		computeMeanPercentage(Percentual, percCounter);

	
	/***    FREE DEVICE MEMORY    ***/
    hipFree(Ddistance);
	hipFree(Dvertex);
    hipFree(DMatrix);
    hipFree(Dedges);

	/***    FREE HOST MEMORY    ***/
	free(Distance);
	free(matrix);
    free(Queue);
}




int main(int argc, char *argv[]){

	/***    CONTROL PARAMETERS NUMBER    ***/
	if(argc < 2)
	{
		printf("\nUsage: ./stConnectivity 'input_file'\n\n");
		return -1;
	}


	/***    READ GRAPH FROM FILE    ***/
	int N, E, nof_lines;
 	GDirection GraphDirection;
 	GraphDirection = UNDIRECTED;  			//DIRECTED = 0, UNDIRECTED = 1, UNDEFINED = 2
 	readGraph::readGraphHeader(argv[1], N, E, nof_lines, GraphDirection);
    Graph graph(N, E, GraphDirection);
    readGraph::readSTD(argv[1], graph, nof_lines);


    /***    PRINT CONFIG INFO    ***/
	std::cout << "\n----------------------KERNEL INFO---------------------" 			<< std::endl
    	 << "            Block dimension : " <<  BLOCK_SIZE 							<< std::endl
    	 << "      Max concurrent blocks : " <<  MAX_CONCURR_BL(BLOCK_SIZE) 			<< std::endl
    	 << "       Shared Memory per SM : " <<  SMem_Per_SM 							<< std::endl
    	 << "    Shared Memory per block : " <<  SMem_Per_Block(BLOCK_SIZE) 			<< std::endl
    	 << "Int Shared Memory per block : " <<  IntSMem_Per_Block(BLOCK_SIZE) 			<< std::endl
    	 << "         Frontier dimension : " <<  FRONTIER_SIZE 							<< std::endl
    	 << "         Int frontier limit : " <<  BLOCK_FRONTIER_LIMIT 					<< std::endl
		 << "--------------------------------------------------------" 			<< std::endl << std::endl;


	/***    LAUNCH ST-CONN FUNCTION    ***/
    if(argc > 2)
    {
    	int Nsources = atoi(argv[2]);
    	printf("Launch stConnectivity with %d sources\n", Nsources);
		doSTCONN(graph, N, E, Nsources);
    }
    else
    {
		for (int i = 0; i < LENGTH; ++i)
		{
			printf("Launch stConnectivity with %d sources\n", parameters[i]);
			doSTCONN(graph, N, E, parameters[i]);
		}
    }

	return 0;
}
