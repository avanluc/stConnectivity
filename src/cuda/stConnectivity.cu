#include "hip/hip_runtime.h"
#pragma once

#include "KernelStConnectivity_WEfficient.cu"
#include "stConn.h"

using namespace std;


int main(int argc, char *argv[]){

	// If wrong number of argumants print usage
	if(argc < 3)
	{
		printf("\nUsage: ./stConnectivity 'input_file' '#_distinguished_nodes'\n\n");
		return -1;
	}


	/***    READ GRAPH FROM FILE    ***/
	int N, E, nof_lines;
 	GDirection GraphDirection;  // scelta dell'utente oppure vuote (direzione di default estratta dal file) valori possibili = DIRECTED, UNDIRECTED
 	readGraph::readGraphHeader(argv[1], N, E, nof_lines, GraphDirection);
    Graph graph(N, E, GraphDirection);
    readGraph::readSTD(argv[1], graph, nof_lines);



	/***    READ GRAPH FROM FILE    ***/
	/*int N, E;
	edge *graph;
	ifstream in (argv[1]);
	in >> N >> E;
	graph = (edge*)malloc(E*sizeof(edge));
	ReadGraph(argv[1], graph, &N, &E);
	
	// Memory allocation and initialization
	int *vertex = (int*)calloc(N+1, sizeof(int));
	int *edges = (int*)calloc(E, sizeof(int));
	
	// Creation of CSR structure
	GraphToCSR(graph, vertex, edges, N, E);

	printf("Transfert graph to CSR structure\n");
	*/
	// int source = atoi(argv[2]);
	// int target = atoi(argv[3]);
	// int Nsources = atoi(argv[4]);


	// calculate size for allocation
	int Nsources = atoi(argv[2]);

	size_t sizeE 	  = E * sizeof(int);
	size_t sizeN 	  = N * sizeof(int2);
	size_t sizeN1 	  = (N+1) * sizeof(int);
	size_t sizeN3 	  = Nsources * sizeof(int);
	size_t sizeMatrix = Nsources * Nsources * sizeof(bool);

	

	/***    ALLOCATE HOST MEMORY    ***/
	int2 *Distance 	= (int2*)calloc(N, sizeof(int2));	
	int *sources 	= (int*)calloc(Nsources, sizeof(int));
	int *Queue 		= (int*)calloc(Nsources, sizeof(int));	
	bool *matrix 	= (bool*)calloc(Nsources * Nsources, sizeof(bool));


	/***    ALLOCATE DEVICE MEMORY    ***/
	int *Dedges;
	int *Dvertex;
	int *Dsources;
	int2 *Ddistance;
	bool *DMatrix;

	gpuErrchk( hipMalloc((void **) &Dvertex, 	sizeN1) );
	gpuErrchk( hipMalloc((void **) &Dedges, 	sizeE) );
	gpuErrchk( hipMalloc((void **) &DMatrix, 	sizeMatrix) );
	gpuErrchk( hipMalloc((void **) &Ddistance, sizeN) );
	gpuErrchk( hipMalloc((void **) &Dsources, 	sizeN3) );
/*
	printf("Device memory allocated\n");

	cout << "\n---------------------KERNEL INFO---------------------" 					<< endl
    	 << "            Block dimension : " <<  BLOCK_SIZE 							<< endl
    	 << "      Max concurrent blocks : " <<  MAX_CONCURR_BL(BLOCK_SIZE) 			<< endl
    	 << "   Number of current blocks : " <<  Nsources 								<< endl
    	 << "       Shared Memory per SM : " <<  SMem_Per_SM 							<< endl
    	 << "    Shared Memory per block : " <<  SMem_Per_Block(BLOCK_SIZE) 			<< endl
    	 << "Int Shared Memory per block : " <<  IntSMem_Per_Block(BLOCK_SIZE) 			<< endl
    	 << "                  F1_OFFSET : " <<  F1_OFFSET 								<< endl
    	 << "               F1 dimension : " <<  SMem_Per_Block(BLOCK_SIZE)-F1_OFFSET	<< endl
    	 << "         Frontier dimension : " <<  FRONTIER_SIZE 							<< endl
    	 << "       Block frontier limit : " <<  BLOCK_FRONTIER_LIMIT 					<< endl
		 << "-------------------------------------------------------" 			<< endl << endl;
*/
 	//vector< ONodes > ordered = OrderNodes(vertex, N);

	/***    SERVICE VARIABLES    ***/
 	vector<double> mean_times(3);
 	vector<double> par_times(N_TEST);
 	vector<double> seq_times(N_TEST);
 	vector<long double> Percentual(N_TEST);
 	int percCounter = 0;


	srand (time(NULL));
 	for (int test = 0; test < N_TEST; test++)
	{

		int source = rand() % N;
		int target = rand() % N;

	    ChooseRandomNodes(sources, graph.OutNodes, N, Nsources, source, target);

	    // inizializzazione dei valori a INT_MAX
	    for (int i = 0; i < N; i++){
	    	Distance[i].x = INT_MAX;
	    	Distance[i].y = INT_MAX;
	    }

	    // inizializzazione dei valori dei nodi distinti a Distance 0 e Color id
	    for (int i = 0; i < Nsources; i++){
			int j = sources[i];
			Distance[j].x = 0;
			Distance[j].y = i;
		}
		
		int VisitedNodes = 0;
	    
	    /***    MEMCOPY HOST_TO_DEVICE    ***/
	    gpuErrchk( hipMemcpy(Dvertex, graph.OutNodes, sizeN1, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dedges, graph.OutEdges, sizeE, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(DMatrix, matrix, sizeMatrix, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Ddistance, Distance, sizeN, hipMemcpyHostToDevice) );
	    gpuErrchk( hipMemcpy(Dsources, sources, sizeN3, hipMemcpyHostToDevice) );
		
		if(ATOMIC)
			gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(GlobalCounter), &VisitedNodes, sizeof(int)) );
	    
		// Allocate CUDA events to be used for timing
	    hipEvent_t start;
	    hipEvent_t start1;
	    hipEvent_t stop;
	    hipEvent_t stop1;

	    gpuErrchk( hipEventCreate(&start) );
	    gpuErrchk( hipEventCreate(&stop) );
	    
	    // Record the start event
	    gpuErrchk( hipEventRecord(start, NULL) );

	    // Launch Cuda Kernel
		dim3 block(BLOCK_SIZE, 1);


		/***    LAUNCH KERNEL    ***/
		// if(SINGLE_BLOCK){
		//     dim3 grid(1, 1);
		//     BFS_BlockKernel<<< grid, block, SMem_Per_SM>>>(Dvertex, Dedges, Dsources, Ddistance, /*Dcolor,*/ DMatrix, Nsources);
		// }
		// else{
		    dim3 grid(Nsources, 1);
		    BFS_BlockKernel<<< grid, block, SMem_Per_Block(BLOCK_SIZE)>>>(Dvertex, Dedges, Dsources, Ddistance, /*Dcolor,*/ DMatrix, Nsources);
		// }

		// gpuErrchk( hipMemcpyFromSymbol(&VisitedNodes, HIP_SYMBOL(GlobalCounter), sizeof(int), 0, hipMemcpyDeviceToHost) );
		// VisitedNodes += Nsources;
		// long double perc = ((long double)VisitedNodes / (long double)N) * 100;
		// //cout << "              Visited Nodes : " << VisitedNodes << endl 
		// //	 << "                Graph Nodes : "<< N << endl;
		// //printf("  Graph Visitage Percentual : %.2Lf%\n", perc);
		// if(VisitedNodes < N){
		// 	printf("---------------WARNING: BFS NOT COMPLETE---------------\t\t\t\t%.2Lf%\n", perc);
		// 	Percentual[percCounter] = perc;
		// 	percCounter++;
		// }
		// else{
		// 	printf("-------------------------------------------------------\n");
		// }

		/*gpuErrchk( hipMemcpy(Distance, Ddistance, sizeN, hipMemcpyDeviceToHost) );
		for (int i = 0; i < N; i++){
	    	if (Distance[i].x == INT_MAX)
	    		printf("---WARNING--- Nodo %d non visitato!!\n", i);
	    }*/

	    /***    MEMCOPY DEVICE_TO_HOST    ***/
	    if(!BFS)
			gpuErrchk( hipMemcpy(matrix, DMatrix, sizeMatrix, hipMemcpyDeviceToHost) );
		

		// for (int i = 0; i < Nsources; ++i)
		// {
		// 	printf("| ");
		// 	for (int j = 0; j < Nsources; ++j)
		// 		printf("%d ", matrix[Nsources*i+j]);
		// 	printf("|\n");
		// }
		//printf("matrix completed\n");
		
	    gpuErrchk( hipEventRecord(stop, NULL) );
	    gpuErrchk( hipEventSynchronize(stop) );
	    float msecTotal = 0.0f;
	    float msecTotal1 = 0.0f;
	    bool connect = false;


	    /***    MATRIX VISIT ON HOST    ***/
	    if(!BFS){
		    Timer<HOST> TM;
		    TM.start();
		    
			connect = MatrixBFS(matrix, Nsources, 0, 1, Queue);

			TM.stop();
	    	
		    msecTotal1 = TM.duration();
	    }


		/***    CALCULATE ELAPSED TIME    ***/
	    gpuErrchk( hipEventElapsedTime(&msecTotal, start, stop) );

	    if(DEBUG){
			printf("#%d:\tsource: %d    \ttarget: %d   \tresult: %c[%d;%dm%s%c[%dm   \tElapsed time = %c[%d;%dm%.1f%c[%dm ms\n", 
															test, source, target, 27, 0, 31 + connect,(connect ? "true" : "false"), 
															27, 0, 27, 0, 31, msecTotal + msecTotal1, 27, 0);
	    }
		par_times[test] = msecTotal;
		seq_times[test] = msecTotal1;
	}
	if(N_TEST > 1)
	{
		double sum_par = 0;
		double sum_seq = 0;
		for (int i = 1; i < N_TEST; ++i){
			sum_par += par_times[i];
			sum_seq += seq_times[i];
		}
		printf("\nN: %d\n", Nsources);
		printf("AVG TIME \t\t: %c[%d;%dm%.1f%c[%dm ms\n", 27, 0, 31, (sum_par + sum_seq) / (N_TEST-1), 27, 0);
		printf("AVG PARALLEL TIME \t: %c[%d;%dm%.1f%c[%dm ms\n", 27, 0, 31, sum_par / (N_TEST-1), 27, 0);
		printf("AVG MATRIX BFS TIME \t: %c[%d;%dm%.1f%c[%dm ms\n\n", 27, 0, 31, sum_seq / (N_TEST-1), 27, 0);
	}

	// double sum = 0;
	// for(int i = 0; i < percCounter; i++){
	// 	sum += Percentual[i];
	// 	printf("sum = %f", sum);
	// }
	// printf("\n\nAVG Percentual \t\t: %.2f%\n", sum / percCounter);
	// printf("MIN Percentual \t\t: %.2f%\n", min(Percentual, percCounter));
	// printf("MAX Percentual \t\t: %.2f%\n", max(Percentual, percCounter));

	
	/***    FREE MEMORY    ***/
	hipFree(Dvertex);
    hipFree(Dedges);
    hipFree(DMatrix);
    hipFree(Ddistance);

    free(Queue);
	free(matrix);
	free(Distance);
	//free(vertex);
	//free(edges);
	//free(graph);

	return 0;
}
