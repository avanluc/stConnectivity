#include "hip/hip_runtime.h"
#pragma once

#include <assert.h>
#include "prefixSumAsm.cu"
#include "GlobalSync.cu"
#include "definition.cuh"

__device__ int GlobalCounter = 0;
__device__ int globalMax = 0;
__device__ int exitFlag = 0;

extern __shared__ unsigned char SMem[];


 /*
* Assert for CUDA functions
*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



/*
* Self made atomic function to store a int2 value 
*/
__device__ __forceinline__ void atomicStore(int2* address, int2 val){
    unsigned long long* addr_as_ull = (unsigned long long*)address;
    unsigned long long  old = *addr_as_ull;
    unsigned long long  assumed;
    assumed = old;
    atomicCAS(addr_as_ull, assumed, *(unsigned long long*)&val);
    return;
}



__device__ __forceinline__ void FrontierReserve_Block(int* Front_size, int founds, int& n, int &totalBlock, int& globalOffset){
	int* SM = (int*) &SMem[TEMP_POS];
	n = founds;
	const int warpId = WarpID();
	SM[warpId] = warpExclusiveScan<32>(n);

	__syncthreads();
	if (Tid < BLOCK_SIZE / 32)
	{
		int sum = SM[Tid];
		const int total = warpExclusiveScan<BLOCK_SIZE / 32>(sum);

		if (Tid == 0)
		{
			SM[32] = total;
			SM[33] = atomicAdd(Front_size, total);
		}
		SM[Tid] = sum;
	}
	__syncthreads();

	n += SM[warpId];
	totalBlock = SM[32];
	globalOffset = SM[33];
}



__device__ __forceinline__ void Write(int* devFrontier, int* Front_size, int* Queue, int founds) {
		
		int n, total, globalOffset;
		FrontierReserve_Block(Front_size, founds, n, total, globalOffset);

		const int pos = globalOffset + n;
		for (int i = 0; i < founds; i++)
		{
			if((pos + i) < BLOCK_FRONTIER_LIMIT)
			{
				devFrontier[pos + i] = Queue[i];
			}
			else if(!ATOMIC)
			{
				//cudaAssert((pos + i) < BLOCK_FRONTIER_LIMIT, (pos + i));
				exitFlag = 1;
			}
		}
}



__device__ __forceinline__ void swapDev(int*& A, int*& B) {
	int* temp = A;
	A = B;
	B = temp;
}



__global__ void BFS_BlockKernel (	const int* __restrict__ devNode,
									const int* __restrict__ devEdge,
									const int* __restrict__ devSource,
									int2* __restrict__ devDistance,
									bool* __restrict__ Matrix,
									const int Nsources) {
	int Queue[REG_QUEUE];
	int FrontierSize = 1;
	int level = 0;

	if(!ATOMIC)
		exitFlag = 0;

	int* SMemF1 = (int*) &SMem[F1_OFFSET];
	int* F2SizePtr = (int*) &SMem[F2Size_POS];

	if (Tid < FrontierSize)
		SMemF1[Tid] = devSource[blockIdx.x]; 

	while (FrontierSize && FrontierSize < BLOCK_FRONTIER_LIMIT )
	{
		if(!ATOMIC && exitFlag)
			break;

		int founds = 0;
		for (int t = Tid; t < FrontierSize; t += BLOCK_SIZE)
		{
			const int index = SMemF1[t];
			const int start = devNode[index];
			const int2 current = devDistance[index];
			int end = devNode[index + 1];	

			for (int k = start; k < end; k++)
			{
				const int dest = devEdge[k];
				const int2 destination = devDistance[dest];	
				if(ATOMIC)
				{	
					int old = atomicCAS(&devDistance[dest].x, INT_MAX, level);
					if ( old == INT_MAX) {	
						devDistance[dest].x = level;
						devDistance[dest].y = current.y;
						Queue[founds++] = dest;
					}
					/* update adj matrix */
					else if (destination.y != current.y && destination.y < Nsources){	
						Matrix[ (current.y * Nsources) + destination.y ] = true;	
						Matrix[ (destination.y * Nsources) + current.y ] = true;	
					}
				}
				else if(BFS)
				{
					if (destination.x == INT_MAX){	
						devDistance[dest].x = level;
						devDistance[dest].y = current.y;
						Queue[founds++] = dest;
					}
				}
				else
				{
					if (destination.x == INT_MAX) {	
						devDistance[dest].x = level;
						devDistance[dest].y = current.y;
						Queue[founds++] = dest;
					}
					else if (destination.y != current.y && destination.y < Nsources){	
						Matrix[ (current.y * Nsources) + destination.y ] = true;	
						Matrix[ (destination.y * Nsources) + current.y ] = true;	
					}
				}
			}
		}
		
		Write(SMemF1, &F2SizePtr[0], Queue, founds);

		level++;

		FrontierSize = F2SizePtr[0];
		if(ATOMIC && Tid == 0)
			atomicAdd(&GlobalCounter, FrontierSize);

		__syncthreads();
		F2SizePtr[0] = 0;
		//if(Tid == 0 && FrontierSize > globalMax);
			//atomicCAS(&globalMax, globalMax, FrontierSize);
	}
}